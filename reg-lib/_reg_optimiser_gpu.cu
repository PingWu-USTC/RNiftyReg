#include "hip/hip_runtime.h"
#ifndef _reg_optimiser_CU
#define _reg_optimiser_CU

#include "_reg_optimiser_gpu.h"
#include "_reg_optimiser_kernels.cu"

/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_optimiser_gpu::reg_optimiser_gpu()
{
    this->dofNumber=0;
    this->ndim=3;
    this->optimiseX=true;
    this->optimiseY=true;
    this->optimiseZ=true;
    this->currentDOF=NULL;
    this->bestDOF=NULL;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_optimiser_gpu::~reg_optimiser_gpu()
{
    if(this->bestDOF!=NULL)
        cudaCommon_free<float4>(&this->bestDOF);;
    this->bestDOF=NULL;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::Initialise(size_t nvox,
                                   int dim,
                                   bool optX,
                                   bool optY,
                                   bool optZ,
                                   size_t maxit,
                                   size_t start,
                                   InterfaceOptimiser *obj,
                                   float4 *cppData,
                                   float4 *gradData
                                   )
{
    this->dofNumber=nvox;
    this->ndim=dim;
    this->optimiseX=optX;
    this->optimiseY=optY;
    this->optimiseZ=optZ;
    this->maxIterationNumber=maxit;
    this->currentIterationNumber=start;

    this->currentDOF=cppData;

    if(gradient!=NULL)
        this->gradient=gradData;

    if(this->bestDOF!=NULL)
        cudaCommon_free<float4>(&this->bestDOF);

    if(cudaCommon_allocateArrayToDevice(&this->bestDOF,
                                        (int)(this->dofNumber/this->ndim))){
        printf("[NiftyReg ERROR] Error when allocating the best control point array on the GPU.\n");
        exit(1);
    }

    reg_optimiser_gpu::StoreCurrentDOF();

    this->objFunc=obj;
    this->bestObjFunctionValue = this->currentObjFunctionValue =
            this->objFunc->GetObjectiveFunctionValue();
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::RestoreBestDOF()
{
    // restore forward transformation
    NR_CUDA_SAFE_CALL(
        hipMemcpy(this->currentDOF,
                   this->bestDOF,
                   this->GetVoxNumber()*sizeof(float4),
                   hipMemcpyDeviceToDevice))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::StoreCurrentDOF()
{
    // save forward transformation
    NR_CUDA_SAFE_CALL(
        hipMemcpy(this->bestDOF,
                   this->currentDOF,
                   this->GetVoxNumber()*sizeof(float4),
                   hipMemcpyDeviceToDevice))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::Perturbation(float length)
{
    /// @todo
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::NormaliseGradient()
{
    // First compute the gradienfloat max length for normalisation purpose
    float maxGradValue=reg_getMaximalLength_gpu(&this->gradient,
                                                (int)(this->dofNumber / this->ndim));
#ifndef NDEBUG
    printf("[NiftyReg DEBUG] Objective function gradient maximal length: %g\n",maxGradValue);
#endif

    reg_multiplyValue_gpu((int)(this->dofNumber / this->ndim),
                          &this->gradient,
                          1.f/maxGradValue);
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_conjugateGradient_gpu::reg_conjugateGradient_gpu()
    :reg_optimiser_gpu::reg_optimiser_gpu()
{
    this->array1=NULL;
    this->array2=NULL;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_conjugateGradient_gpu::~reg_conjugateGradient_gpu()
{
    if(this->array1!=NULL)
        cudaCommon_free<float4>(&this->array1);
    this->array1=NULL;

    if(this->array2!=NULL)
        cudaCommon_free<float4>(&this->array2);
    this->array2=NULL;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::Initialise(size_t nvox,
                                           int dim,
                                           bool optX,
                                           bool optY,
                                           bool optZ,
                                           size_t maxit,
                                           size_t start,
                                           InterfaceOptimiser *obj,
                                           float4 *cppData,
                                           float4 *gradData
                                           )
{
    reg_optimiser_gpu::Initialise(nvox,
                                  dim,
                                  optX,
                                  optY,
                                  optZ,
                                  maxit,
                                  start,
                                  obj,
                                  cppData,
                                  gradData
                                  );
    this->firstcall=true;
    reg_optimiser_gpu *super = reinterpret_cast<reg_optimiser_gpu *>(this);
    if(cudaCommon_allocateArrayToDevice(&this->array1,
                                        (int)(super->GetVoxNumber()))){
        printf("[NiftyReg ERROR] Error when allocating the first conjugate gradient array on the GPU.\n");
        exit(1);
    }
    if(cudaCommon_allocateArrayToDevice(&this->array2,
                                        (int)(this->GetVoxNumber()))){
        printf("[NiftyReg ERROR] Error when allocating the second conjugate gradient array on the GPU.\n");
        exit(1);
    }
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::UpdateGradientValues()
{
    if(this->firstcall==true){
        reg_initialiseConjugateGradient(&this->gradient,
                                        &this->array1,
                                        &this->array2,
                                        (int)(this->GetVoxNumber()));
        this->firstcall=false;
    }
    else{
        reg_GetConjugateGradient(&this->gradient,
                                 &this->array1,
                                 &this->array2,
                                 (int)(this->GetVoxNumber()));
    }
    return;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::Optimise(float maxLength,
                                 float smallLength,
                                 float &startLength)
{
    size_t lineIteration=0;
    float addedLength=0;
    float currentLength=startLength;

    this->NormaliseGradient();

    // Start performing the line search
    while(currentLength>smallLength &&
          lineIteration<12 &&
          this->currentIterationNumber<this->maxIterationNumber){

        // Compute the gradient normalisation value
        float normValue = -currentLength;

        this->objFunc->UpdateParameters(normValue);

        // Compute the new value
        this->currentObjFunctionValue=this->objFunc->GetObjectiveFunctionValue();

        // Check if the update lead to an improvement of the objective function
        if(this->currentObjFunctionValue > this->bestObjFunctionValue){
#ifndef NDEBUG
            printf("[NiftyReg DEBUG] [%i] objective function: %g | Increment %g | ACCEPTED\n",
                   (int)this->currentIterationNumber,
                   this->currentObjFunctionValue,
                   currentLength);
#endif
            // Improvement - Save the new objective function value
            this->bestObjFunctionValue=this->currentObjFunctionValue;
            // Update the total added length
            addedLength += currentLength;
            // Increase the step size
            currentLength *= 1.1f;
            currentLength = (currentLength<maxLength)?currentLength:maxLength;
            // Save the current deformation parametrisation
            this->StoreCurrentDOF();
        }
        else{
#ifndef NDEBUG
            printf("[NiftyReg DEBUG] [%i] objective function: %g | Increment %g | REJECTED\n",
                   (int)this->currentIterationNumber,
                   this->currentObjFunctionValue,
                   currentLength);
#endif
            // No improvement - Decrease the step size
            currentLength*=0.5;
        }
        this->IncrementCurrentIterationNumber();
        ++lineIteration;
    }
    // update the current size for the next iteration
    startLength=addedLength;
    // Restore the last best deformation parametrisation
    this->RestoreBestDOF();
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::Optimise(float maxLength,
                                         float smallLength,
                                         float &startLength)
{
    this->UpdateGradientValues();
    reg_optimiser_gpu::Optimise(maxLength,
                                smallLength,
                                startLength);
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::Perturbation(float length)
{
    reg_optimiser_gpu::Perturbation(length);
    this->firstcall=true;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_initialiseConjugateGradient(float4 **nodeNMIGradientArray_d,
                                     float4 **conjugateG_d,
                                     float4 **conjugateH_d,
                                     int nodeNumber)
{
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
            NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)))

            const unsigned int Grid_reg_initialiseConjugateGradient =
            (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_initialiseConjugateGradient));
    dim3 G1(Grid_reg_initialiseConjugateGradient,Grid_reg_initialiseConjugateGradient,1);
    dim3 B1(Block_reg_initialiseConjugateGradient,1,1);

    reg_initialiseConjugateGradient_kernel <<< G1, B1 >>> (*conjugateG_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
            NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))
            NR_CUDA_SAFE_CALL(hipMemcpy(*conjugateH_d, *conjugateG_d, nodeNumber*sizeof(float4), hipMemcpyDeviceToDevice))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_GetConjugateGradient(float4 **nodeNMIGradientArray_d,
                              float4 **conjugateG_d,
                              float4 **conjugateH_d,
                              int nodeNumber)
{
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
            NR_CUDA_SAFE_CALL(hipBindTexture(0, conjugateGTexture, *conjugateG_d, nodeNumber*sizeof(float4)))
            NR_CUDA_SAFE_CALL(hipBindTexture(0, conjugateHTexture, *conjugateH_d, nodeNumber*sizeof(float4)))
            NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)))

            // gam = sum((grad+g)*grad)/sum(HxG);
            const unsigned int Grid_reg_GetConjugateGradient1 = (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_GetConjugateGradient1));
    dim3 B1(Block_reg_GetConjugateGradient1,1,1);
    dim3 G1(Grid_reg_GetConjugateGradient1,Grid_reg_GetConjugateGradient1,1);

    float2 *sum_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&sum_d, nodeNumber*sizeof(float2)))
            reg_GetConjugateGradient1_kernel <<< G1, B1 >>> (sum_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
            float2 *sum_h;NR_CUDA_SAFE_CALL(hipHostMalloc(&sum_h, nodeNumber*sizeof(float2)))
            NR_CUDA_SAFE_CALL(hipMemcpy(sum_h,sum_d, nodeNumber*sizeof(float2),hipMemcpyDeviceToHost))
            NR_CUDA_SAFE_CALL(hipFree(sum_d))
            double dgg = 0.0;
    double gg = 0.0;
    for(int i=0; i<nodeNumber; i++){
        dgg += sum_h[i].x;
        gg += sum_h[i].y;
    }
    float gam = (float)(dgg / gg);
    NR_CUDA_SAFE_CALL(hipHostFree((void *)sum_h))

            NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ScalingFactor),&gam,sizeof(float)))
            const unsigned int Grid_reg_GetConjugateGradient2 = (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_GetConjugateGradient2));
    dim3 B2(Block_reg_GetConjugateGradient2,1,1);
    dim3 G2(Grid_reg_GetConjugateGradient2,Grid_reg_GetConjugateGradient2,1);
    reg_GetConjugateGradient2_kernel <<< G2, B2 >>> (*nodeNMIGradientArray_d, *conjugateG_d, *conjugateH_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)

            NR_CUDA_SAFE_CALL(hipUnbindTexture(conjugateGTexture))
            NR_CUDA_SAFE_CALL(hipUnbindTexture(conjugateHTexture))
            NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))

}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
float reg_getMaximalLength_gpu(	float4 **nodeNMIGradientArray_d,
                               int nodeNumber)
{

    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
            NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)))

            // each thread extract the maximal value out of 128
            const int threadNumber = (int)ceil((float)nodeNumber/128.0f);
    const unsigned int Grid_reg_getMaximalLength = (unsigned int)ceil(sqrtf((float)threadNumber/(float)Block_reg_getMaximalLength));
    dim3 B1(Block_reg_getMaximalLength,1,1);
    dim3 G1(Grid_reg_getMaximalLength,Grid_reg_getMaximalLength,1);

    float *all_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&all_d, threadNumber*sizeof(float)))
            reg_getMaximalLength_kernel <<< G1, B1 >>> (all_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)

            float *all_h;NR_CUDA_SAFE_CALL(hipHostMalloc(&all_h, nodeNumber*sizeof(float)))
            NR_CUDA_SAFE_CALL(hipMemcpy(all_h, all_d, threadNumber*sizeof(float),hipMemcpyDeviceToHost))
            NR_CUDA_SAFE_CALL(hipFree(all_d))
            double maxDistance = 0.0f;
    for(int i=0; i<threadNumber; i++) maxDistance = all_h[i]>maxDistance?all_h[i]:maxDistance;
    NR_CUDA_SAFE_CALL(hipHostFree((void *)all_h))

            NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))
            return (float)maxDistance;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_updateControlPointPosition_gpu(nifti_image *controlPointImage,
                                        float4 **controlPointImageArray_d,
                                        float4 **bestControlPointPosition_d,
                                        float4 **nodeNMIGradientArray_d,
                                        float currentLength)
{
    const int nodeNumber = controlPointImage->nx * controlPointImage->ny * controlPointImage->nz;
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
            NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ScalingFactor),&currentLength,sizeof(float)))

            NR_CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *bestControlPointPosition_d, nodeNumber*sizeof(float4)))
            NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)))

            const unsigned int Grid_reg_updateControlPointPosition = (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_updateControlPointPosition));
    dim3 B1(Block_reg_updateControlPointPosition,1,1);
    dim3 G1(Grid_reg_updateControlPointPosition,Grid_reg_updateControlPointPosition,1);

    reg_updateControlPointPosition_kernel <<< G1, B1 >>> (*controlPointImageArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
            NR_CUDA_SAFE_CALL(hipUnbindTexture(controlPointTexture))
            NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */

#endif // _reg_optimiser_CU
