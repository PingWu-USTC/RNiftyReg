#include "hip/hip_runtime.h"
#ifndef _reg_optimiser_CU
#define _reg_optimiser_CU

#include "_reg_optimiser_gpu.h"
#include "_reg_optimiser_kernels.cu"

/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_optimiser_gpu::reg_optimiser_gpu()
    :reg_optimiser<float>::reg_optimiser<float>()
{
    this->dofNumber=0;
    this->ndim=3;
    this->optimiseX=true;
    this->optimiseY=true;
    this->optimiseZ=true;
    this->currentDOF_gpu=NULL;
    this->bestDOF_gpu=NULL;
    this->gradient_gpu=NULL;
#ifndef NDEBUG
    printf("[NiftyReg DEBUG] reg_optimiser_gpu::reg_optimiser() called\n");
#endif
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_optimiser_gpu::~reg_optimiser_gpu()
{
    if(this->bestDOF_gpu!=NULL)
        cudaCommon_free<float4>(&this->bestDOF_gpu);;
    this->bestDOF_gpu=NULL;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::Initialise(size_t nvox,
                                   int dim,
                                   bool optX,
                                   bool optY,
                                   bool optZ,
                                   size_t maxit,
                                   size_t start,
                                   InterfaceOptimiser *obj,
                                   float *cppData,
                                   float *gradData,
                                   size_t a,
                                   float *b,
                                   float *c
                                   )
{
    this->dofNumber=nvox;
    this->ndim=dim;
    this->optimiseX=optX;
    this->optimiseY=optY;
    this->optimiseZ=optZ;
    this->maxIterationNumber=maxit;
    this->currentIterationNumber=start;

    this->currentDOF_gpu=reinterpret_cast<float4 *>(cppData);

    if(gradData!=NULL)
        this->gradient_gpu=reinterpret_cast<float4 *>(gradData);

    if(this->bestDOF_gpu!=NULL)
        cudaCommon_free<float4>(&this->bestDOF_gpu);

    if(cudaCommon_allocateArrayToDevice(&this->bestDOF_gpu,
                                        (int)(this->dofNumber/this->ndim))){
        printf("[NiftyReg ERROR] Error when allocating the best control point array on the GPU.\n");
        exit(1);
    }

    reg_optimiser_gpu::StoreCurrentDOF();

    this->objFunc=obj;
    this->bestObjFunctionValue = this->currentObjFunctionValue =
            this->objFunc->GetObjectiveFunctionValue();
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::RestoreBestDOF()
{
    // restore forward transformation
    NR_CUDA_SAFE_CALL(
        hipMemcpy(this->currentDOF_gpu,
                   this->bestDOF_gpu,
                   this->GetVoxNumber()*sizeof(float4),
                   hipMemcpyDeviceToDevice))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::StoreCurrentDOF()
{
    // save forward transformation
    NR_CUDA_SAFE_CALL(
        hipMemcpy(this->bestDOF_gpu,
                   this->currentDOF_gpu,
                   this->GetVoxNumber()*sizeof(float4),
                   hipMemcpyDeviceToDevice))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::Perturbation(float length)
{
    /// @todo
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::NormaliseGradient()
{
    // First compute the gradienfloat max length for normalisation purpose
    float maxGradValue=reg_getMaximalLength_gpu(&this->gradient_gpu,
                                                (int)(this->dofNumber / this->ndim));
#ifndef NDEBUG
    printf("[NiftyReg DEBUG] Objective function gradient_gpu maximal length: %g\n",maxGradValue);
#endif

    reg_multiplyValue_gpu((int)(this->dofNumber / this->ndim),
                          &this->gradient_gpu,
                          1.f/maxGradValue);
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_conjugateGradient_gpu::reg_conjugateGradient_gpu()
    :reg_optimiser_gpu::reg_optimiser_gpu()
{
    this->array1=NULL;
    this->array2=NULL;
#ifndef NDEBUG
    printf("[NiftyReg DEBUG] reg_conjugateGradient_gpu::reg_optimiser() called\n");
#endif
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
reg_conjugateGradient_gpu::~reg_conjugateGradient_gpu()
{
    if(this->array1!=NULL)
        cudaCommon_free<float4>(&this->array1);
    this->array1=NULL;

    if(this->array2!=NULL)
        cudaCommon_free<float4>(&this->array2);
    this->array2=NULL;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::Initialise(size_t nvox,
                                               int dim,
                                               bool optX,
                                               bool optY,
                                               bool optZ,
                                               size_t maxit,
                                               size_t start,
                                               InterfaceOptimiser *obj,
                                               float *cppData,
                                               float *gradData,
                                               size_t a,
                                               float *b,
                                               float *c)
{
    reg_optimiser_gpu::Initialise(nvox,
                                  dim,
                                  optX,
                                  optY,
                                  optZ,
                                  maxit,
                                  start,
                                  obj,
                                  cppData,
                                  gradData
                                  );
    this->firstcall=true;
    reg_optimiser_gpu *super = reinterpret_cast<reg_optimiser_gpu *>(this);
    if(cudaCommon_allocateArrayToDevice(&this->array1,
                                        (int)(super->GetVoxNumber()))){
        printf("[NiftyReg ERROR] Error when allocating the first conjugate gradient_gpu array on the GPU.\n");
        exit(1);
    }
    if(cudaCommon_allocateArrayToDevice(&this->array2,
                                        (int)(this->GetVoxNumber()))){
        printf("[NiftyReg ERROR] Error when allocating the second conjugate gradient_gpu array on the GPU.\n");
        exit(1);
    }
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::UpdateGradientValues()
{
    if(this->firstcall==true){
        reg_initialiseConjugateGradient_gpu(&(this->gradient_gpu),
                                            &(this->array1),
                                            &(this->array2),
                                            (int)(this->GetVoxNumber()));
        this->firstcall=false;
    }
    else{
        reg_GetConjugateGradient_gpu(&this->gradient_gpu,
                                     &this->array1,
                                     &this->array2,
                                     (int)(this->GetVoxNumber()));
    }
    return;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_optimiser_gpu::Optimise(float maxLength,
                                 float smallLength,
                                 float &startLength)
{
    size_t lineIteration=0;
    float addedLength=0;
    float currentLength=startLength;

    // Start performing the line search
    while(currentLength>smallLength &&
          lineIteration<12 &&
          this->currentIterationNumber<this->maxIterationNumber){

        // Compute the gradient_gpu normalisation value
        float normValue = -currentLength;

        this->objFunc->UpdateParameters(normValue);

        // Compute the new value
        this->currentObjFunctionValue=this->objFunc->GetObjectiveFunctionValue();

        // Check if the update lead to an improvement of the objective function
        if(this->currentObjFunctionValue > this->bestObjFunctionValue){
#ifndef NDEBUG
            printf("[NiftyReg DEBUG] [%i] objective function: %g | Increment %g | ACCEPTED\n",
                   (int)this->currentIterationNumber,
                   this->currentObjFunctionValue,
                   currentLength);
#endif
            // Improvement - Save the new objective function value
            this->bestObjFunctionValue=this->currentObjFunctionValue;
            // Update the total added length
            addedLength += currentLength;
            // Increase the step size
            currentLength *= 1.1f;
            currentLength = (currentLength<maxLength)?currentLength:maxLength;
            // Save the current deformation parametrisation
            this->StoreCurrentDOF();
        }
        else{
#ifndef NDEBUG
            printf("[NiftyReg DEBUG] [%i] objective function: %g | Increment %g | REJECTED\n",
                   (int)this->currentIterationNumber,
                   this->currentObjFunctionValue,
                   currentLength);
#endif
            // No improvement - Decrease the step size
            currentLength*=0.5;
        }
        this->IncrementCurrentIterationNumber();
        ++lineIteration;
    }
    // update the current size for the next iteration
    startLength=addedLength;
    // Restore the last best deformation parametrisation
    this->RestoreBestDOF();
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::Optimise(float maxLength,
                                         float smallLength,
                                         float &startLength)
{
    this->UpdateGradientValues();
    reg_optimiser_gpu::Optimise(maxLength,
                                smallLength,
                                startLength);
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_conjugateGradient_gpu::Perturbation(float length)
{
    reg_optimiser_gpu::Perturbation(length);
    this->firstcall=true;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_initialiseConjugateGradient_gpu(float4 **nodeGradientArray_d,
                                         float4 **conjugateG_d,
                                         float4 **conjugateH_d,
                                         int nodeNumber)
{
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeGradientArray_d, nodeNumber*sizeof(float4)))

    const unsigned int Grid_reg_initialiseConjugateGradient =
    (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_initialiseConjugateGradient));
    dim3 G1(Grid_reg_initialiseConjugateGradient,Grid_reg_initialiseConjugateGradient,1);
    dim3 B1(Block_reg_initialiseConjugateGradient,1,1);

    reg_initialiseConjugateGradient_kernel <<< G1, B1 >>> (*conjugateG_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))
    NR_CUDA_SAFE_CALL(hipMemcpy(*conjugateH_d, *conjugateG_d, nodeNumber*sizeof(float4), hipMemcpyDeviceToDevice))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_GetConjugateGradient_gpu(float4 **nodeGradientArray_d,
                                  float4 **conjugateG_d,
                                  float4 **conjugateH_d,
                                  int nodeNumber)
{
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, conjugateGTexture, *conjugateG_d, nodeNumber*sizeof(float4)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, conjugateHTexture, *conjugateH_d, nodeNumber*sizeof(float4)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeGradientArray_d, nodeNumber*sizeof(float4)))

    // gam = sum((grad+g)*grad)/sum(HxG);
    const unsigned int Grid_reg_GetConjugateGradient1 = (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_GetConjugateGradient1));
    dim3 B1(Block_reg_GetConjugateGradient1,1,1);
    dim3 G1(Grid_reg_GetConjugateGradient1,Grid_reg_GetConjugateGradient1,1);

    float2 *sum_d;
    NR_CUDA_SAFE_CALL(hipMalloc(&sum_d, nodeNumber*sizeof(float2)))
    reg_GetConjugateGradient1_kernel <<< G1, B1 >>> (sum_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    float2 *sum_h;NR_CUDA_SAFE_CALL(hipHostMalloc(&sum_h, nodeNumber*sizeof(float2)))
    NR_CUDA_SAFE_CALL(hipMemcpy(sum_h,sum_d, nodeNumber*sizeof(float2),hipMemcpyDeviceToHost))
    NR_CUDA_SAFE_CALL(hipFree(sum_d))
    double dgg = 0.0;
    double gg = 0.0;
    for(int i=0; i<nodeNumber; i++){
    dgg += sum_h[i].x;
    gg += sum_h[i].y;
    }
    float gam = (float)(dgg / gg);
    NR_CUDA_SAFE_CALL(hipHostFree((void *)sum_h))

    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ScalingFactor),&gam,sizeof(float)))
    const unsigned int Grid_reg_GetConjugateGradient2 = (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_GetConjugateGradient2));
    dim3 B2(Block_reg_GetConjugateGradient2,1,1);
    dim3 G2(Grid_reg_GetConjugateGradient2,Grid_reg_GetConjugateGradient2,1);
    reg_GetConjugateGradient2_kernel <<< G2, B2 >>> (*nodeGradientArray_d, *conjugateG_d, *conjugateH_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)

    NR_CUDA_SAFE_CALL(hipUnbindTexture(conjugateGTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(conjugateHTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))

}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
float reg_getMaximalLength_gpu(float4 **nodeGradientArray_d,
                               int nodeNumber)
{
    // Copy constant memory value and bind texture
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeGradientArray_d, nodeNumber*sizeof(float4)))

    float *dist_d=NULL;
    NR_CUDA_SAFE_CALL(hipMalloc(&dist_d,nodeNumber*sizeof(float)))

    const unsigned int Grid_reg_getEuclideanDistance = (unsigned int)ceil(sqrtf((float)nodeNumber/(float)512));
    dim3 B1(512,1,1);
    dim3 G1(Grid_reg_getEuclideanDistance,Grid_reg_getEuclideanDistance,1);
    reg_getEuclideanDistance_kernel <<< G1, B1 >>> (dist_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))

    float maxDistance = reg_maxReduction_gpu(dist_d,nodeNumber);
    NR_CUDA_SAFE_CALL(hipFree(dist_d))

    return maxDistance;
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
void reg_updateControlPointPosition_gpu(nifti_image *controlPointImage,
                                        float4 **controlPointImageArray_d,
                                        float4 **bestControlPointPosition_d,
                                        float4 **nodeGradientArray_d,
                                        float currentLength)
{
    const int nodeNumber = controlPointImage->nx * controlPointImage->ny * controlPointImage->nz;
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)))
    NR_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ScalingFactor),&currentLength,sizeof(float)))

    NR_CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *bestControlPointPosition_d, nodeNumber*sizeof(float4)))
    NR_CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeGradientArray_d, nodeNumber*sizeof(float4)))

    const unsigned int Grid_reg_updateControlPointPosition =
            (unsigned int)ceil(sqrtf((float)nodeNumber/(float)Block_reg_updateControlPointPosition));
    dim3 B1(Block_reg_updateControlPointPosition,1,1);
    dim3 G1(Grid_reg_updateControlPointPosition,Grid_reg_updateControlPointPosition,1);

    reg_updateControlPointPosition_kernel <<< G1, B1 >>> (*controlPointImageArray_d);
    NR_CUDA_CHECK_KERNEL(G1,B1)
    NR_CUDA_SAFE_CALL(hipUnbindTexture(controlPointTexture))
    NR_CUDA_SAFE_CALL(hipUnbindTexture(gradientImageTexture))
}
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */
/* \/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/\/ */

#endif // _reg_optimiser_CU
